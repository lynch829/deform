#include "hip/hip_runtime.h"
#include "gpu/cost_function.h"
#include "hybrid_graph_cut_optimizer.h"

#include <stk/cuda/hip/hip_runtime.h>
#include <stk/cuda/stream.h>
#include <stk/cuda/volume.h>

namespace cuda = stk::cuda;

__global__ void apply_displacement_delta_kernel(
    cuda::VolumePtr<float4> df,
    cuda::VolumePtr<uint8_t> labels,
    dim3 dims,
    float4 delta
)
{
    int x = blockIdx.x*blockDim.x + threadIdx.x;
    int y = blockIdx.y*blockDim.y + threadIdx.y;
    int z = blockIdx.z*blockDim.z + threadIdx.z;

    if (x >= dims.x ||
        y >= dims.y ||
        z >= dims.z)
    {
        return;
    }

    df(x,y,z) = df(x,y,z) + delta * labels(x,y,z);
}

void HybridGraphCutOptimizer::apply_displacement_delta(
    const float3& delta,
    stk::GpuVolume& df,
    cuda::Stream& stream
)
{
    dim3 dims = df.size();
    dim3 block_size {32,32,1};
    dim3 grid_size {
        (dims.x + block_size.x - 1) / block_size.x,
        (dims.y + block_size.y - 1) / block_size.y,
        (dims.z + block_size.z - 1) / block_size.z
    };

    apply_displacement_delta_kernel<<<grid_size, block_size, 0, stream>>>(
        df, 
        _gpu_labels, 
        dims, 
        float4{delta.x, delta.y, delta.z, 0.0f}
    );
    CUDA_CHECK_ERRORS(hipPeekAtLastError());
}

__global__ void reduce_total_energy(
    cuda::VolumePtr<float2> unary_term,
    cuda::VolumePtr<float4> binary_term_x, // Regularization cost in x+
    cuda::VolumePtr<float4> binary_term_y, // y+
    cuda::VolumePtr<float4> binary_term_z, // z+
    dim3 dims,
    float* out
)
{
    extern __shared__ float shared[];

    int x = blockIdx.x*blockDim.x + threadIdx.x;
    int y = blockIdx.y*blockDim.y + threadIdx.y;
    int z = blockIdx.z*blockDim.z + threadIdx.z;

    int tid = threadIdx.x + threadIdx.y * blockDim.x + threadIdx.z * blockDim.x * blockDim.y;
    int bid = blockIdx.x + blockIdx.y * gridDim.x + blockIdx.z * gridDim.x * gridDim.y;

    shared[tid] = 0;

    if (x < dims.x &&
        y < dims.y &&
        z < dims.z)
    {
        float e = unary_term(x,y,z).x;
        if (x + 1 < int(dims.x)) {
            e += binary_term_x(x,y,z).x;
        }
        if (y + 1 < int(dims.y)) {
            e += binary_term_y(x,y,z).x;
        }
        if (z + 1 < int(dims.z)) {
            e += binary_term_z(x,y,z).x;
        }

        shared[tid] = e;
    }
    __syncthreads();

    #define REDUCTION_STEP(n_) \
        if (tid < (n_)) shared[tid] = shared[tid] + shared[tid+(n_)]; \
        __syncthreads();

    REDUCTION_STEP(512);
    REDUCTION_STEP(256);
    REDUCTION_STEP(128);
    REDUCTION_STEP(64);
    REDUCTION_STEP(32);
    REDUCTION_STEP(16);
    REDUCTION_STEP(8);
    REDUCTION_STEP(4);
    REDUCTION_STEP(2);

    if (tid == 0) {
        out[bid] = shared[0] + shared[1];
    }

    #undef REDUCTION_STEP
}

double HybridGraphCutOptimizer::calculate_energy(
    GpuUnaryFunction& unary_fn,
    GpuBinaryFunction& binary_fn,
    stk::GpuVolume& df
)
{
    reset_unary_cost();

    dim3 dims = _gpu_unary_cost.size();
    int3 begin {0, 0, 0};
    int3 end {(int)dims.x, (int)dims.y, (int)dims.z};

    cuda::Stream& stream = stk::cuda::Stream::null();
    unary_fn(df, {0,0,0}, begin, end, _gpu_unary_cost, stream);
    
    // Compute binary terms
    binary_fn(
        df,
        {0, 0, 0},
        begin,
        end,
        _gpu_binary_cost_x,
        _gpu_binary_cost_y,
        _gpu_binary_cost_z,
        stream
    );

    dim3 block_size{32,32,1};

    dim3 grid_size {
        (dims.x + block_size.x - 1) / block_size.x,
        (dims.y + block_size.y - 1) / block_size.y,
        (dims.z + block_size.z - 1) / block_size.z
    };
    uint32_t n_blocks = grid_size.x * grid_size.y * grid_size.z;

    float* d_block_sum;
    CUDA_CHECK_ERRORS(hipMalloc(&d_block_sum, n_blocks*sizeof(float)));

    reduce_total_energy<<<grid_size, block_size, 
        uint32_t(sizeof(float)*1024)>>>
    (
        _gpu_unary_cost,
        _gpu_binary_cost_x,
        _gpu_binary_cost_y,
        _gpu_binary_cost_z,
        dims,
        d_block_sum
    );

    CUDA_CHECK_ERRORS(hipPeekAtLastError());
    CUDA_CHECK_ERRORS(hipDeviceSynchronize());

    float* block_sum = new float[n_blocks];
    CUDA_CHECK_ERRORS(hipMemcpy(block_sum, d_block_sum, n_blocks*sizeof(float), hipMemcpyDeviceToHost));

    // TODO: Perform all reduction on GPU
    double total_energy = 0;
    for (int i = 0; i < (int)n_blocks; ++i) {
        total_energy += block_sum[i];
    }

    delete [] block_sum;
    CUDA_CHECK_ERRORS(hipFree(d_block_sum));

    return total_energy;
}
