#include "hip/hip_runtime.h"
#include "gpu/cost_functions/cost_function.h"
#include "hybrid_graph_cut_optimizer.h"

#include <stk/cuda/hip/hip_runtime.h>
#include <stk/cuda/stream.h>
#include <stk/cuda/volume.h>

namespace cuda = stk::cuda;

__global__ void apply_displacement_delta_additive_kernel(
    cuda::VolumePtr<float4> df,
    cuda::VolumePtr<uint8_t> labels,
    dim3 dims,
    float4 delta
)
{
    int x = blockIdx.x*blockDim.x + threadIdx.x;
    int y = blockIdx.y*blockDim.y + threadIdx.y;
    int z = blockIdx.z*blockDim.z + threadIdx.z;

    if (x >= dims.x ||
        y >= dims.y ||
        z >= dims.z)
    {
        return;
    }

    df(x,y,z) = df(x,y,z) + delta * labels(x,y,z);
}

__global__ void apply_displacement_delta_compositive_kernel(
    cuda::VolumePtr<float4> df_in,
    cuda::VolumePtr<float4> df_out,
    cuda::VolumePtr<uint8_t> labels,
    dim3 dims,
    float4 delta
)
{
    int x = blockIdx.x*blockDim.x + threadIdx.x;
    int y = blockIdx.y*blockDim.y + threadIdx.y;
    int z = blockIdx.z*blockDim.z + threadIdx.z;

    if (x >= dims.x ||
        y >= dims.y ||
        z >= dims.z)
    {
        return;
    }

    auto const lac = cuda::linear_at_clamp<float4>;

    df_out(x,y,z) = lac(df_in, dims, 
        x + delta.x * labels(x,y,z),
        y + delta.y * labels(x,y,z),
        z + delta.z * labels(x,y,z)
    ) + delta * labels(x,y,z);
}

void apply_displacement_delta(
    stk::GpuVolume& df_in,
    stk::GpuVolume& df_out,
    stk::GpuVolume& labels,
    const float3& delta,
    Settings::UpdateRule update_rule,
    cuda::Stream stream
)
{
    dim3 dims = df_in.size();
    dim3 block_size {32,32,1};
    dim3 grid_size {
        (dims.x + block_size.x - 1) / block_size.x,
        (dims.y + block_size.y - 1) / block_size.y,
        (dims.z + block_size.z - 1) / block_size.z
    };

    if (update_rule == Settings::UpdateRule_Additive) {
        apply_displacement_delta_additive_kernel
        <<<grid_size, block_size, 0, stream>>>(
            df_in,
            df_out,
            labels,
            dims,
            float4{delta.x, delta.y, delta.z, 0.0f}
        );
    }
    else if (update_rule == Settings::UpdateRule_Compositve) {
        apply_displacement_delta_compositive_kernel
        <<<grid_size, block_size, 0, stream>>>(
            df_in,
            df_out,
            labels,
            dims,
            float4{delta.x, delta.y, delta.z, 0.0f}
        );
    }
    CUDA_CHECK_ERRORS(hipPeekAtLastError());
}

__global__ void reduce_total_energy(
    cuda::VolumePtr<float2> unary_term,
    cuda::VolumePtr<float4> binary_term_x, // Regularization cost in x+
    cuda::VolumePtr<float4> binary_term_y, // y+
    cuda::VolumePtr<float4> binary_term_z, // z+
    dim3 dims,
    float* out
)
{
    extern __shared__ float shared[];

    int x = blockIdx.x*blockDim.x + threadIdx.x;
    int y = blockIdx.y*blockDim.y + threadIdx.y;
    int z = blockIdx.z*blockDim.z + threadIdx.z;

    int tid = threadIdx.x + threadIdx.y * blockDim.x + threadIdx.z * blockDim.x * blockDim.y;
    int bid = blockIdx.x + blockIdx.y * gridDim.x + blockIdx.z * gridDim.x * gridDim.y;

    shared[tid] = 0;

    if (x < dims.x &&
        y < dims.y &&
        z < dims.z)
    {
        float e = unary_term(x,y,z).x;
        if (x + 1 < int(dims.x)) {
            e += binary_term_x(x,y,z).x;
        }
        if (y + 1 < int(dims.y)) {
            e += binary_term_y(x,y,z).x;
        }
        if (z + 1 < int(dims.z)) {
            e += binary_term_z(x,y,z).x;
        }

        shared[tid] = e;
    }
    __syncthreads();

    #define REDUCTION_STEP(n_) \
        if (tid < (n_)) shared[tid] = shared[tid] + shared[tid+(n_)]; \
        __syncthreads();

    REDUCTION_STEP(512);
    REDUCTION_STEP(256);
    REDUCTION_STEP(128);
    REDUCTION_STEP(64);
    REDUCTION_STEP(32);
    REDUCTION_STEP(16);
    REDUCTION_STEP(8);
    REDUCTION_STEP(4);
    REDUCTION_STEP(2);

    if (tid == 0) {
        out[bid] = shared[0] + shared[1];
    }

    #undef REDUCTION_STEP
}

double calculate_energy(
    GpuUnaryFunction& unary_fn,
    GpuBinaryFunction& binary_fn,
    stk::GpuVolume& df,
    stk::GpuVolume& unary_cost,
    stk::GpuVolume& binary_cost_x,
    stk::GpuVolume& binary_cost_y,
    stk::GpuVolume& binary_cost_z
)
{
    // Reset unary cost
    hipExtent extent = make_hipExtent(
        unary_cost.size().x * sizeof(float2),
        unary_cost.size().y,
        unary_cost.size().z
    );
    CUDA_CHECK_ERRORS(hipMemset3D(unary_cost.pitched_ptr(), 0, extent));

    dim3 dims = unary_cost.size();
    int3 begin {0, 0, 0};
    int3 end {(int)dims.x, (int)dims.y, (int)dims.z};

    cuda::Stream& stream = stk::cuda::Stream::null();
    unary_fn(df, {0,0,0}, begin, end, unary_cost, stream);

    // Compute binary terms
    binary_fn(
        df,
        {0, 0, 0},
        begin,
        end,
        binary_cost_x,
        binary_cost_y,
        binary_cost_z,
        stream
    );

    dim3 block_size{32,32,1};

    dim3 grid_size {
        (dims.x + block_size.x - 1) / block_size.x,
        (dims.y + block_size.y - 1) / block_size.y,
        (dims.z + block_size.z - 1) / block_size.z
    };
    uint32_t n_blocks = grid_size.x * grid_size.y * grid_size.z;

    float* d_block_sum;
    CUDA_CHECK_ERRORS(hipMalloc(&d_block_sum, n_blocks*sizeof(float)));

    reduce_total_energy<<<grid_size, block_size,
        uint32_t(sizeof(float)*1024)>>>
    (
        unary_cost,
        binary_cost_x,
        binary_cost_y,
        binary_cost_z,
        dims,
        d_block_sum
    );

    CUDA_CHECK_ERRORS(hipPeekAtLastError());
    CUDA_CHECK_ERRORS(hipDeviceSynchronize());

    float* block_sum = new float[n_blocks];
    CUDA_CHECK_ERRORS(hipMemcpy(block_sum, d_block_sum, n_blocks*sizeof(float), hipMemcpyDeviceToHost));

    // TODO: Perform all reduction on GPU
    double total_energy = 0;
    for (int i = 0; i < (int)n_blocks; ++i) {
        total_energy += block_sum[i];
    }

    delete [] block_sum;
    CUDA_CHECK_ERRORS(hipFree(d_block_sum));

    return total_energy;
}
