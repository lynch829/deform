#include "hip/hip_runtime.h"
#include "resample.h"

#include <stk/common/assert.h>
#include <stk/cuda/hip/hip_runtime.h>
#include <stk/cuda/ptr.h>
#include <stk/image/gpu_volume.h>
#include <stk/math/float3.h>
#include <stk/math/float4.h>

#include "gaussian_filter.h"

#include <algorithm>

#ifdef DF_ENABLE_DISPLACEMENT_FIELD_RESIDUALS
    #error Displacement residuals not implemented for CUDA
#endif

namespace cuda = stk::cuda;

template<typename T>
__global__ void shrink_volume_by_2_kernel(
    const cuda::VolumePtr<T> in,
    dim3 new_dims,
    cuda::VolumePtr<T> out
)
{
    int x = blockIdx.x*blockDim.x + threadIdx.x;
    int y = blockIdx.y*blockDim.y + threadIdx.y;
    int z = blockIdx.z*blockDim.z + threadIdx.z;

    if (x >= new_dims.x ||
        y >= new_dims.y ||
        z >= new_dims.z)
    {
        return;
    }

    out(x, y, z) = in(int(2*x), int(2*y), int(2*z));
}

__global__ void downsample_vectorfield_kernel(
    const cuda::VolumePtr<float4> field,
    dim3 old_dims,
    dim3 new_dims,
    float inv_scale,
    cuda::VolumePtr<float4> out
)
{
    int x = blockIdx.x*blockDim.x + threadIdx.x;
    int y = blockIdx.y*blockDim.y + threadIdx.y;
    int z = blockIdx.z*blockDim.z + threadIdx.z;

    if (x >= new_dims.x ||
        y >= new_dims.y ||
        z >= new_dims.z)
    {
        return;
    }

    int px = int(x * inv_scale);
    int py = int(y * inv_scale);
    int pz = int(z * inv_scale);

    float4 v = field(px, py, pz);

    int px1 = min(px+1, old_dims.x-1);
    int py1 = min(py+1, old_dims.y-1);
    int pz1 = min(pz+1, old_dims.z-1);

    v = v + field(px1, py, pz);
    v = v + field(px, py1, pz);
    v = v + field(px, py, pz1);
    v = v + field(px1, py1, pz);
    v = v + field(px1, py, pz1);
    v = v + field(px, py1, pz1);
    v = v + field(px1, py1, pz1);
    
    float s = 1.0f / 8.0f;
    out(x, y, z) = float4{s*v.x, s*v.y, s*v.z, 0.0f};
}



__global__ void upsample_vectorfield_kernel(
    hipTextureObject_t src,
    dim3 new_dims,
    float4 inv_scale,
    cuda::VolumePtr<float4> out
)
{
    int x = blockIdx.x*blockDim.x + threadIdx.x;
    int y = blockIdx.y*blockDim.y + threadIdx.y;
    int z = blockIdx.z*blockDim.z + threadIdx.z;

    if (x >= new_dims.x ||
        y >= new_dims.y ||
        z >= new_dims.z)
    {
        return;
    }

    out(x, y, z) = tex3D<float4>(src, x * inv_scale.x, y * inv_scale.y, z * inv_scale.z);
}

namespace {

/// Shrinks the volume by removing every other element
stk::GpuVolume shrink_volume_by_2(const stk::GpuVolume& src)
{
    ASSERT(src.voxel_type() == stk::Type_Float ||
           src.voxel_type() == stk::Type_Float4);
    ASSERT(src.usage() == stk::gpu::Usage_PitchedPointer);

    dim3 old_dims = src.size();
    dim3 new_dims {
        uint32_t(ceil(old_dims.x * 0.5f)),
        uint32_t(ceil(old_dims.y * 0.5f)),
        uint32_t(ceil(old_dims.z * 0.5f)),
    };

    stk::GpuVolume dest(new_dims, src.voxel_type());
    dest.copy_meta_from(src);
    
    float3 old_spacing = src.spacing();
    float3 new_spacing {
        old_spacing.x * (old_dims.x / float(new_dims.x)),
        old_spacing.y * (old_dims.y / float(new_dims.y)),
        old_spacing.z * (old_dims.z / float(new_dims.z))
    };
    dest.set_spacing(new_spacing);

    dim3 block_size{8,8,1};
    dim3 grid_size {
        (new_dims.x + block_size.x - 1) / block_size.x,
        (new_dims.y + block_size.y - 1) / block_size.y,
        (new_dims.z + block_size.z - 1) / block_size.z
    };

    if (src.voxel_type() == stk::Type_Float) {
        shrink_volume_by_2_kernel<float><<<grid_size, block_size>>>(
            src.pitched_ptr(),
            new_dims,
            dest.pitched_ptr()
        );
    }
    else if (src.voxel_type() == stk::Type_Float4) {
        shrink_volume_by_2_kernel<float4><<<grid_size, block_size>>>(
            src.pitched_ptr(),
            new_dims,
            dest.pitched_ptr()
        );
    }
    else {
        ASSERT(false);
    }
    CUDA_CHECK_ERRORS(hipDeviceSynchronize());

    return dest;
}

}


namespace filters {
namespace gpu {
    stk::GpuVolume downsample_volume_by_2(const stk::GpuVolume& vol)
    {
        ASSERT(vol.voxel_type() == stk::Type_Float);
        ASSERT(vol.usage() == stk::gpu::Usage_PitchedPointer);
    
        float3 spacing = vol.spacing();
        float unit_sigma = std::min(spacing.x, std::min(spacing.y, spacing.z));

        stk::GpuVolume filtered = gaussian_filter_3d(vol, unit_sigma);

        return shrink_volume_by_2(filtered);
    }

    stk::GpuVolume downsample_vectorfield_by_2(const stk::GpuVolume& vol
#ifdef DF_ENABLE_DISPLACEMENT_FIELD_RESIDUALS
    , stk::GpuVolume& residual
#endif
    )
    {
        ASSERT(vol.voxel_type() == stk::Type_Float4);
        ASSERT(vol.usage() == stk::gpu::Usage_PitchedPointer);
    
        float3 spacing = vol.spacing();
        float unit_sigma = std::min(spacing.x, std::min(spacing.y, spacing.z));

        stk::GpuVolume filtered = gaussian_filter_3d(vol, unit_sigma);

        return shrink_volume_by_2(filtered);
    }

    stk::GpuVolume upsample_vectorfield(const stk::GpuVolume& vol, const dim3& new_dims
#ifdef DF_ENABLE_DISPLACEMENT_FIELD_RESIDUALS
    , const stk::GpuVolume& residual
#endif
    )
    {
        ASSERT(vol.voxel_type() == stk::Type_Float4); // No float3 in gpu volumes
        ASSERT(vol.usage() == stk::gpu::Usage_PitchedPointer);
        
        dim3 old_dims = vol.size();
        float3 inv_scale{
            float(old_dims.x) / new_dims.x,
            float(old_dims.y) / new_dims.y,
            float(old_dims.z) / new_dims.z
        };
        
        // TODO: Any extra cost of BindAsSurface (for returned volume)
        stk::GpuVolume out(vol.size(), vol.voxel_type());
        out.copy_meta_from(vol);

        float3 old_spacing = vol.spacing();
        float3 new_spacing{
            old_spacing.x * inv_scale.x,
            old_spacing.y * inv_scale.y,
            old_spacing.z * inv_scale.z
        };
        out.set_spacing(new_spacing);

        // hipResourceDesc res_desc;
        // memset(&res_desc, 0, sizeof(res_desc));
        // res_desc.resType = hipResourceTypePitch2D;
        // res_desc.array = vol.pitched_ptr();
        
        // hipTextureDesc tex_desc{0};
        // tex_desc.addressMode[0] = hipAddressModeClamp;
        // tex_desc.addressMode[1] = hipAddressModeClamp;
        // tex_desc.addressMode[2] = hipAddressModeClamp;
        // tex_desc.filterMode = hipFilterModeLinear;

        // hipTextureObject_t src_obj{0};
        // hipCreateTextureObject(&src_obj, &res_desc, &tex_desc, nullptr);

        // dim3 block_size{8,8,1};
        // dim3 grid_size {
        //     (new_dims.x + block_size.x - 1) / block_size.x,
        //     (new_dims.y + block_size.y - 1) / block_size.y,
        //     (new_dims.z + block_size.z - 1) / block_size.z
        // };

        // upsample_vectorfield_kernel<<<grid_size, block_size>>>(
        //     src_obj,
        //     new_dims,
        //     inv_scale,
        //     out
        // );

        return out;
    }
}
}